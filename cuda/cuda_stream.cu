#include <cstdio>
#include <cstdlib>
#include <iostream>
using namespace std;

#include <hip/hip_runtime.h>

#define CUDA_CALL(func, name)						\
  {									\
    hipError_t e = (func);						\
    if(e != hipSuccess)						\
      cout << "CUDA: " << hipGetErrorString(e) << ": " << name << endl; \
    else								\
      cout << "CUDA SUCC: " << (name) << endl;				\
  }

void fill_array(int * data, const int num) {
  for(int i = 0; i < num; i++){
    data[i] = i;
  }
}

void check_array(char * device_prefix,
		 int * data,
		 const int num) {
  bool error_found = false;
  for(int i = 0; i < num; i++) {
    if(data[i] != i * 2){
      cout << "error: " << device_prefix << "\t" << i << "\t" << data[i] << endl;
      error_found = true;
    }
  }
  if (!error_found)
    cout << "passed: " << device_prefix << endl;
}

__global__ void gpu_test_kernel(int * data) {
  const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  for(int i = 0; i < 10000; i++){
    data[tid] *= 2;
    data[tid] /= 2;
  }
  data[tid] *= 2;
}

#define MAX_NUM_DEVICES (4)
#define NUM_ELEM (1024*1024*8)

hipStream_t stream[MAX_NUM_DEVICES];

char device_prefix[MAX_NUM_DEVICES][300];

int * gpu_data[MAX_NUM_DEVICES];
int * cpu_src_data[MAX_NUM_DEVICES];
int * cpu_dst_data[MAX_NUM_DEVICES];

hipEvent_t kernel_start_event[MAX_NUM_DEVICES];
hipEvent_t memcpy_to_start_event[MAX_NUM_DEVICES];
hipEvent_t memcpy_from_start_event[MAX_NUM_DEVICES];
hipEvent_t memcpy_from_stop_event[MAX_NUM_DEVICES];

__host__ void gpu_kernel(void) {

  const int shared_memory_usage = 0;
  const size_t single_gpu_chunk_size = sizeof(int) * NUM_ELEM;
  const int num_threads = 256;
  const int num_blocks = ((NUM_ELEM + (num_threads - 1)) / num_threads);
  cout << "begin" << endl;

  int num_devices;
  CUDA_CALL(hipGetDeviceCount(&num_devices), "hipGetDeviceCount");
  if(num_devices > MAX_NUM_DEVICES)
    num_devices = MAX_NUM_DEVICES;
  
  cout << "num devices: " << num_devices << endl;

  for(int device_num = 0; device_num < num_devices; device_num++) {
    CUDA_CALL(hipSetDevice(device_num), "hipSetDevice");
    
    struct hipDeviceProp_t device_prop;
    CUDA_CALL(hipGetDeviceProperties(&device_prop,
				      device_num), "hipGetDeviceProperties");
    sprintf(&device_prefix[device_num][0], "\nID: %d %s : ", device_num,
	    device_prop.name);
    
    CUDA_CALL(hipStreamCreate(&stream[device_num]), "hipStreamCreate");
    CUDA_CALL(hipMalloc((void**)&gpu_data[device_num], single_gpu_chunk_size), "hipMalloc");

    CUDA_CALL(hipHostMalloc((void**)&cpu_src_data[device_num],
			     single_gpu_chunk_size, hipHostMallocDefault), "hipHostMalloc");

    CUDA_CALL(hipHostMalloc((void**)&cpu_dst_data[device_num],
			     single_gpu_chunk_size, hipHostMallocDefault), "hipHostMalloc");

    fill_array(cpu_src_data[device_num], NUM_ELEM);

    CUDA_CALL(hipEventCreate(&memcpy_to_start_event[device_num]), "create memcpy_to_start_event");
    CUDA_CALL(hipEventCreate(&kernel_start_event[device_num]), "create kernel_start_event");
    CUDA_CALL(hipEventCreate(&memcpy_from_start_event[device_num]), "create memcpy_from_start_event");
    CUDA_CALL(hipEventCreate(&memcpy_from_stop_event[device_num]), "create memcpy_from_stop_event");

    CUDA_CALL(hipEventRecord(memcpy_to_start_event[device_num]), "memcpy_to_start_event");
    CUDA_CALL(hipMemcpyAsync(gpu_data[device_num],
			      cpu_src_data[device_num],
			      single_gpu_chunk_size,
			      hipMemcpyHostToDevice,
			      stream[device_num]), "hipMemcpyAsync");

    CUDA_CALL(hipEventRecord(kernel_start_event[device_num]), "hipEventRecord");
    gpu_test_kernel<<<num_blocks, 
      num_threads, 
      shared_memory_usage,
      stream[device_num]>>>(gpu_data[device_num]);

    CUDA_CALL(hipEventRecord(memcpy_from_start_event[device_num]), "memcpy_from_start_event");
    CUDA_CALL(hipMemcpyAsync(cpu_dst_data[device_num],
			      gpu_data[device_num],
			      single_gpu_chunk_size,
			      hipMemcpyDeviceToHost,
			      stream[device_num]), "hipMemcpyAsync");
    CUDA_CALL(hipEventRecord(memcpy_from_stop_event[device_num]), "memcpy_from_stop_event");
  }
  
  for(int device_num = 0; device_num < num_devices; device_num++) {
    CUDA_CALL(hipSetDevice(device_num), "");
    CUDA_CALL(hipStreamSynchronize(stream[device_num]), "");

    float time_copy_to_ms;
    CUDA_CALL(hipEventElapsedTime(&time_copy_to_ms,
				   memcpy_to_start_event[device_num],
				   kernel_start_event[device_num]), "");

    float time_kernel_ms;
    CUDA_CALL(hipEventElapsedTime(&time_kernel_ms,
				   kernel_start_event[device_num],
				   memcpy_from_start_event[device_num]), "");

    float time_copy_from_ms;
    CUDA_CALL(hipEventElapsedTime(&time_copy_from_ms,
				   memcpy_from_start_event[device_num],
				   memcpy_from_stop_event[device_num]), "");

    float time_exec_ms;
    CUDA_CALL(hipEventElapsedTime(&time_exec_ms,
				   memcpy_to_start_event[device_num],
				   memcpy_from_stop_event[device_num]), "");    

    CUDA_CALL(hipStreamDestroy(stream[device_num]), "");
    CUDA_CALL(hipFree(gpu_data[device_num]), "");

    check_array(device_prefix[device_num],
		cpu_dst_data[device_num],
		NUM_ELEM);

    CUDA_CALL(hipHostFree(cpu_src_data[device_num]), "");
    CUDA_CALL(hipHostFree(cpu_dst_data[device_num]), "");
    CUDA_CALL(hipDeviceReset(), "");

    cout << time_copy_to_ms << "\t" << time_kernel_ms << "\t" << time_copy_from_ms << "\n"
	 << time_exec_ms << "\t" << time_copy_to_ms + time_kernel_ms + time_copy_from_ms << endl;
  }
}

int main(){
  gpu_kernel();
}